#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <cutil.h>
#include "ftocmacros.h"
#define TRUE 1
#define FALSE 0


//!ACC$REGION (<nx>,<ny>) BEGIN
__global__ void loop_ij_Kernel1(int its,int nvl,int npp,int nd,int nx,int ny,int *nprox_ij,float *sidevec_e_ij,float *ue_ij,float *ve_ij,float *vnorm_ij) {

 int ivl;
 int j;
 int i;
 int isn;
  for (ivl=1;ivl<=nvl;ivl++) {
//!ACC$DO VECTOR
    j = threadIdx.x+1;
//    for (j=1;j<=nx;j++) {
//!ACC$DO PARALLEL
      i = blockIdx.x+1;
//      for (i=1;i<=ny;i++) {
        for (isn=1;isn<=nprox_ij[FTNREF2D(i,j,nx,1,1)];isn++) {
          vnorm_ij[FTNREF4D(i,j,isn,ivl,nx,ny,npp,1,1,1,1)] = sidevec_e_ij[FTNREF4D(i,j,2,isn,nx,ny,nd,1,1,1,1)] * ue_ij[FTNREF4D(i,j,isn,ivl,nx,ny,npp,1,1,1,1)] - sidevec_e_ij[FTNREF4D(i,j,1,isn,nx,ny,nd,1,1,1,1)] * ve_ij[FTNREF4D(i,j,isn,ivl,nx,ny,npp,1,1,1,1)];
        }
//      }
//    }
  }
  return;
}
//!ACC$REGION END


extern "C" void loop_ij_ (int *its__G,int *nvl__G,int *npp__G,int *nd__G,int *nx__G,int *ny__G,int *nprox_ij,float *sidevec_e_ij,float *ue_ij,float *ve_ij,float *vnorm_ij) {

  int argc=2;
  char *argv[]= {"","-device=0"};
  CUT_DEVICE_INIT(argc, argv);

  int its=*its__G;
  int nvl=*nvl__G;
  int npp=*npp__G;
  int nd=*nd__G;
  int nx=*nx__G;
  int ny=*ny__G;




  dim3 cuda_threads1(nx);
  dim3 cuda_grids1(ny);

// create and start timer
  unsigned int timer = 0;
  int *d_nprox_ij;
  hipMalloc((void **) &d_nprox_ij,((nx)*(ny))*sizeof(int));
  hipMemcpy(d_nprox_ij,nprox_ij,((nx)*(ny))*sizeof(int),hipMemcpyHostToDevice);
  float *d_sidevec_e_ij;
  hipMalloc((void **) &d_sidevec_e_ij,((nx)*(ny)*(nd)*(npp))*sizeof(float));
  hipMemcpy(d_sidevec_e_ij,sidevec_e_ij,((nx)*(ny)*(nd)*(npp))*sizeof(float),hipMemcpyHostToDevice);
  float *d_ue_ij;
  hipMalloc((void **) &d_ue_ij,((nx)*(ny)*(npp)*(nvl))*sizeof(float));
  hipMemcpy(d_ue_ij,ue_ij,((nx)*(ny)*(npp)*(nvl))*sizeof(float),hipMemcpyHostToDevice);
  float *d_ve_ij;
  hipMalloc((void **) &d_ve_ij,((nx)*(ny)*(npp)*(nvl))*sizeof(float));
  hipMemcpy(d_ve_ij,ve_ij,((nx)*(ny)*(npp)*(nvl))*sizeof(float),hipMemcpyHostToDevice);
  float *d_vnorm_ij;
  hipMalloc((void **) &d_vnorm_ij,((nx)*(ny)*(npp)*(nvl))*sizeof(float));
  hipMemcpy(d_vnorm_ij,vnorm_ij,((nx)*(ny)*(npp)*(nvl))*sizeof(float),hipMemcpyHostToDevice);

  cutCreateTimer(&timer);
  cutStartTimer(timer);
  // setup execution parameters 
  // execute the kernel 
  loop_ij_Kernel1<<< cuda_grids1, cuda_threads1 >>>(its,nvl,npp,nd,nx,ny,d_nprox_ij,d_sidevec_e_ij,d_ue_ij,d_ve_ij,d_vnorm_ij);
  hipDeviceSynchronize();
  // check if kernel execution generated an error
  CUT_CHECK_ERROR("Kernel execution failed");

  cutStopTimer(timer);
  float timer__G = cutGetTimerValue(timer);
  cutDeleteTimer(timer);
  printf("loop_ij_Kernel1 execution time: %f \n",timer__G);
  hipFree(d_nprox_ij);
  hipFree(d_sidevec_e_ij);
  hipFree(d_ue_ij);
  hipFree(d_ve_ij);
  hipMemcpy(vnorm_ij,d_vnorm_ij,((nx)*(ny)*(npp)*(nvl))*sizeof(float),hipMemcpyDeviceToHost);
  hipFree(d_vnorm_ij);


  *its__G=its;
  *nvl__G=nvl;
  *npp__G=npp;
  *nd__G=nd;
  *nx__G=nx;
  *ny__G=ny;

  return;
}
